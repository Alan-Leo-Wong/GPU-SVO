#include "hip/hip_runtime.h"
﻿#include "IO.h"
#include "SVO.h"
#include "CUDAUtil.h"
#include "MortonLUT.h"
#include "libmorton\morton.h"
#include <thrust/scan.h>
#include <thrust/unique.h>
#include <thrust/extrema.h>
#include <thrust/device_vector.h>
#include <hip/hip_cooperative_groups.h>
#include <crt/hip/device_functions.h>
namespace cg = cooperative_groups;

//thrust::device_vector<size_t> d_morton2Idx; // 存储某一层morton code到nodeArray下标的映射
//thrust::device_vector<thrust::device_vector<size_t>> d_allMorton2Idx; // 存储所有层morton code到nodeArray下标的映射
//thrust::device_vector<thrust::device_vector<SVONode>> d_allSVONodeArray; // save all sparse octree nodes

// Estimate best block and grid size using CUDA Occupancy Calculator
int blockSize;   // The launch configurator returned block size 
int minGridSize; // The minimum grid size needed to achieve the maximum occupancy for a full device launch 
int gridSize;    // The actual grid size needed, based on input size 

// 右移三位，并让最高标志位为 1 (同时使之前右移三位后的标志位为0)即可
CUDA_CALLABLE_MEMBER uint32_t getParentMorton(const uint32_t morton)
{
	return (((morton >> 3) & 0xfffffff));
}

CUDA_CALLABLE_MEMBER bool isSameParent(const uint32_t morton_1, const uint32_t morton_2)
{
	return getParentMorton(morton_1) == getParentMorton(morton_2);
}

template <typename T>
struct scanMortonFlag : public thrust::unary_function<T, T> {
	__host__ __device__ T operator()(const T& x) {
		// printf("%lu %d\n", b, (b >> 31) & 1);
		return (x >> 31) & 1;
	}
};

__global__ void surfaceVoxelize(const int nTris,
	const Eigen::Vector3i* d_surfaceVoxelGridSize,
	const Eigen::Vector3f* d_gridOrigin,
	const Eigen::Vector3f* d_unitVoxelSize,
	float* d_triangle_data,
	uint32_t* d_voxelArray)
{
	size_t tid = threadIdx.x + blockIdx.x * blockDim.x;
	size_t stride = blockDim.x * gridDim.x;
	int t_tid = tid;

	const Eigen::Vector3i surfaceVoxelGridSize = *d_surfaceVoxelGridSize;
	const Eigen::Vector3f unitVoxelSize = *d_unitVoxelSize;
	const Eigen::Vector3f gridOrigin = *d_gridOrigin;
	Eigen::Vector3f delta_p{ unitVoxelSize.x(), unitVoxelSize.y(), unitVoxelSize.z() };
	Eigen::Vector3i grid_max{ surfaceVoxelGridSize.x() - 1, surfaceVoxelGridSize.y() - 1, surfaceVoxelGridSize.z() - 1 }; // grid max (grid runs from 0 to gridsize-1)
	while (tid < nTris) { // every thread works on specific triangles in its stride
		size_t t = tid * 9; // triangle contains 9 vertices

		// COMPUTE COMMON TRIANGLE PROPERTIES
		// Move vertices to origin using modelBBox
		Eigen::Vector3f v0 = Eigen::Vector3f(d_triangle_data[t], d_triangle_data[t + 1], d_triangle_data[t + 2]) - gridOrigin;
		Eigen::Vector3f v1 = Eigen::Vector3f(d_triangle_data[t + 3], d_triangle_data[t + 4], d_triangle_data[t + 5]) - gridOrigin;
		Eigen::Vector3f v2 = Eigen::Vector3f(d_triangle_data[t + 6], d_triangle_data[t + 7], d_triangle_data[t + 8]) - gridOrigin;
		// Edge vectors
		Eigen::Vector3f e0 = v1 - v0;
		Eigen::Vector3f e1 = v2 - v1;
		Eigen::Vector3f e2 = v0 - v2;
		// Normal vector pointing up from the triangle
		Eigen::Vector3f n = e0.cross(e1).normalized();

		// COMPUTE TRIANGLE BBOX IN GRID
		// Triangle bounding box in world coordinates is min(v0,v1,v2) and max(v0,v1,v2)
		AABox<Eigen::Vector3f> t_bbox_world(fminf(v0, fminf(v1, v2)), fmaxf(v0, fmaxf(v1, v2)));
		// Triangle bounding box in voxel grid coordinates is the world bounding box divided by the grid unit vector
		AABox<Eigen::Vector3i> t_bbox_grid;
		t_bbox_grid.min = clamp(
			Eigen::Vector3i((t_bbox_world.min.x() / unitVoxelSize.x()), (t_bbox_world.min.y() / unitVoxelSize.y()), (t_bbox_world.min.z() / unitVoxelSize.z())),
			Eigen::Vector3i(0, 0, 0), grid_max
		);
		t_bbox_grid.max = clamp(
			Eigen::Vector3i((t_bbox_world.max.x() / unitVoxelSize.x()), (t_bbox_world.max.y() / unitVoxelSize.y()), (t_bbox_world.max.z() / unitVoxelSize.z())),
			Eigen::Vector3i(0, 0, 0), grid_max
		);

		// PREPARE PLANE TEST PROPERTIES
		Eigen::Vector3f c(0.0f, 0.0f, 0.0f);
		if (n.x() > 0.0f) { c.x() = unitVoxelSize.x(); }
		if (n.y() > 0.0f) { c.y() = unitVoxelSize.y(); }
		if (n.z() > 0.0f) { c.z() = unitVoxelSize.z(); }
		float d1 = n.dot((c - v0));
		float d2 = n.dot(((delta_p - c) - v0));

		// PREPARE PROJECTION TEST PROPERTIES
		// XY plane
		Eigen::Vector2f n_xy_e0(-1.0f * e0.y(), e0.x());
		Eigen::Vector2f n_xy_e1(-1.0f * e1.y(), e1.x());
		Eigen::Vector2f n_xy_e2(-1.0f * e2.y(), e2.x());
		if (n.z() < 0.0f)
		{
			n_xy_e0 = -n_xy_e0;
			n_xy_e1 = -n_xy_e1;
			n_xy_e2 = -n_xy_e2;
		}
		float d_xy_e0 = (-1.0f * n_xy_e0.dot(Eigen::Vector2f(v0.x(), v0.y()))) + fmaxf(0.0f, unitVoxelSize.x() * n_xy_e0[0]) + fmaxf(0.0f, unitVoxelSize.y() * n_xy_e0[1]);
		float d_xy_e1 = (-1.0f * n_xy_e1.dot(Eigen::Vector2f(v1.x(), v1.y()))) + fmaxf(0.0f, unitVoxelSize.x() * n_xy_e1[0]) + fmaxf(0.0f, unitVoxelSize.y() * n_xy_e1[1]);
		float d_xy_e2 = (-1.0f * n_xy_e2.dot(Eigen::Vector2f(v2.x(), v2.y()))) + fmaxf(0.0f, unitVoxelSize.x() * n_xy_e2[0]) + fmaxf(0.0f, unitVoxelSize.y() * n_xy_e2[1]);
		// YZ plane
		Eigen::Vector2f n_yz_e0(-1.0f * e0.z(), e0.y());
		Eigen::Vector2f n_yz_e1(-1.0f * e1.z(), e1.y());
		Eigen::Vector2f n_yz_e2(-1.0f * e2.z(), e2.y());
		if (n.x() < 0.0f) {
			n_yz_e0 = -n_yz_e0;
			n_yz_e1 = -n_yz_e1;
			n_yz_e2 = -n_yz_e2;
		}
		float d_yz_e0 = (-1.0f * n_yz_e0.dot(Eigen::Vector2f(v0.y(), v0.z()))) + fmaxf(0.0f, unitVoxelSize.y() * n_yz_e0[0]) + fmaxf(0.0f, unitVoxelSize.z() * n_yz_e0[1]);
		float d_yz_e1 = (-1.0f * n_yz_e1.dot(Eigen::Vector2f(v1.y(), v1.z()))) + fmaxf(0.0f, unitVoxelSize.y() * n_yz_e1[0]) + fmaxf(0.0f, unitVoxelSize.z() * n_yz_e1[1]);
		float d_yz_e2 = (-1.0f * n_yz_e2.dot(Eigen::Vector2f(v2.y(), v2.z()))) + fmaxf(0.0f, unitVoxelSize.y() * n_yz_e2[0]) + fmaxf(0.0f, unitVoxelSize.z() * n_yz_e2[1]);
		// ZX plane																							 													  
		Eigen::Vector2f n_zx_e0(-1.0f * e0.x(), e0.z());
		Eigen::Vector2f n_zx_e1(-1.0f * e1.x(), e1.z());
		Eigen::Vector2f n_zx_e2(-1.0f * e2.x(), e2.z());
		if (n.y() < 0.0f) {
			n_zx_e0 = -n_zx_e0;
			n_zx_e1 = -n_zx_e1;
			n_zx_e2 = -n_zx_e2;
		}
		float d_xz_e0 = (-1.0f * n_zx_e0.dot(Eigen::Vector2f(v0.z(), v0.x()))) + fmaxf(0.0f, unitVoxelSize.z() * n_zx_e0[0]) + fmaxf(0.0f, unitVoxelSize.x() * n_zx_e0[1]);
		float d_xz_e1 = (-1.0f * n_zx_e1.dot(Eigen::Vector2f(v1.z(), v1.x()))) + fmaxf(0.0f, unitVoxelSize.z() * n_zx_e1[0]) + fmaxf(0.0f, unitVoxelSize.x() * n_zx_e1[1]);
		float d_xz_e2 = (-1.0f * n_zx_e2.dot(Eigen::Vector2f(v2.z(), v2.x()))) + fmaxf(0.0f, unitVoxelSize.z() * n_zx_e2[0]) + fmaxf(0.0f, unitVoxelSize.x() * n_zx_e2[1]);

		// test possible grid boxes for overlap
		for (uint16_t z = t_bbox_grid.min.z(); z <= t_bbox_grid.max.z(); z++) {
			for (uint16_t y = t_bbox_grid.min.y(); y <= t_bbox_grid.max.y(); y++) {
				for (uint16_t x = t_bbox_grid.min.x(); x <= t_bbox_grid.max.x(); x++) {
					// if (checkBit(voxel_table, location)){ continue; }
					// TRIANGLE PLANE THROUGH BOX TEST
					Eigen::Vector3f p(x * unitVoxelSize.x(), y * unitVoxelSize.y(), z * unitVoxelSize.z());
					float nDOTp = n.dot(p);
					if ((nDOTp + d1) * (nDOTp + d2) > 0.0f) { continue; }

					// PROJECTION TESTS
					// XY
					Eigen::Vector2f p_xy(p.x(), p.y());
					if ((n_xy_e0.dot(p_xy) + d_xy_e0) < 0.0f) { continue; }
					if ((n_xy_e1.dot(p_xy) + d_xy_e1) < 0.0f) { continue; }
					if ((n_xy_e2.dot(p_xy) + d_xy_e2) < 0.0f) { continue; }

					// YZ
					Eigen::Vector2f p_yz(p.y(), p.z());
					if ((n_yz_e0.dot(p_yz) + d_yz_e0) < 0.0f) { continue; }
					if ((n_yz_e1.dot(p_yz) + d_yz_e1) < 0.0f) { continue; }
					if ((n_yz_e2.dot(p_yz) + d_yz_e2) < 0.0f) { continue; }

					// XZ	
					Eigen::Vector2f p_zx(p.z(), p.x());
					if ((n_zx_e0.dot(p_zx) + d_xz_e0) < 0.0f) { continue; }
					if ((n_zx_e1.dot(p_zx) + d_xz_e1) < 0.0f) { continue; }
					if ((n_zx_e2.dot(p_zx) + d_xz_e2) < 0.0f) { continue; }

					//size_t mortonCode = mortonEncode_LUT(x, y, z);
					uint32_t mortonCode = mortonEncode_LUT(x, y, z);
					/*if (mortonCode == 63)
					{
						printf("x = %d, y = %d, z = %d, OK!\n", (int)x, (int)y, (int)z);
					}*/
					atomicExch(d_voxelArray + mortonCode, mortonCode | E_MORTON_32_FLAG); // 最高位设置为1，代表这是个表面的voxel
				}
			}
		}
		tid += stride;
	}
}

void SparseVoxelOctree::meshVoxelize(const Eigen::Vector3i* d_surfaceVoxelGridSize,
	const Eigen::Vector3f* d_unitVoxelSize,
	const Eigen::Vector3f* d_gridOrigin,
	thrust::device_vector<uint32_t>& d_CNodeMortonArray)
{
	thrust::device_vector<Eigen::Vector3f> d_triangleThrustVec;
	for (int i = 0; i < nModelTris; ++i)
	{
		d_triangleThrustVec.push_back(modelPoints[idx2Points[i].x()]);
		d_triangleThrustVec.push_back(modelPoints[idx2Points[i].y()]);
		d_triangleThrustVec.push_back(modelPoints[idx2Points[i].z()]);
	}
	float* d_triangleData = (float*)thrust::raw_pointer_cast(&(d_triangleThrustVec[0]));
	getOccupancyMaxPotentialBlockSize(nModelTris, minGridSize, blockSize, gridSize, surfaceVoxelize, 0, 0);
	surfaceVoxelize << <gridSize, blockSize >> > (nModelTris, d_surfaceVoxelGridSize,
		d_gridOrigin, d_unitVoxelSize, d_triangleData, d_CNodeMortonArray.data().get());
	getLastCudaError("Kernel 'surfaceVoxelize' launch failed!\n");
	hipDeviceSynchronize();
}

__global__ void compactArray(const int n,
	const bool* d_isValidArray,
	const uint32_t* d_dataArray,
	const size_t* d_esumDataArray,
	uint32_t* d_pactDataArray)
{
	size_t tid = threadIdx.x + blockIdx.x * blockDim.x;

	if (tid < n && d_isValidArray[tid])
		d_pactDataArray[d_esumDataArray[tid]] = d_dataArray[tid];
}

// 计算表面voxel共对应多少个八叉树节点同时设置父节点的莫顿码数组
__global__ void cpNumNodes(const size_t n,
	const uint32_t* d_pactDataArray,
	size_t* d_nNodesArray,
	uint32_t* d_parentMortonArray)
{
	size_t tid = threadIdx.x + blockIdx.x * blockDim.x;

	if (tid >= 1 && tid < n)
	{
		if (isSameParent(d_pactDataArray[tid], d_pactDataArray[tid - 1])) d_nNodesArray[tid] = 0;
		else
		{
			const uint32_t parentMorton = getParentMorton(d_pactDataArray[tid]);
			d_parentMortonArray[parentMorton] = parentMorton | E_MORTON_32_FLAG;
			d_nNodesArray[tid] = 8;
		}
	}
}

// 根据d_sumNodesArray和d_pactVoxelArray(存储了莫顿码)设置节点数组，本质也是compact
// blockSize必须是32的整数倍，因为最底层节点个数是8的整数倍
//__global__ void createNode(const size_t nNodes,
//	const size_t pactSize,
//	const size_t* d_sumNodesArray,
//	const uint32_t* d_pactDataArray,
//	const Eigen::Vector3f* d_gridOrigin,
//	const float* d_width,
//	SVONode* d_nodeArray/*,
//	size_t* d_morton2Idx*/)
//{
//	extern __shared__ uint32_t sh_nodeMorton[]; // blockSize
//
//	//cg::thread_block ctb = cg::this_thread_block();
//	//cg::thread_group tile8 = cg::tiled_partition(ctb, 8);
//
//	size_t tid = threadIdx.x + blockIdx.x * blockDim.x;
//
//	const Eigen::Vector3f gridOrigin = *d_gridOrigin;
//	const float width = *d_width;
//	/*sh_nodeMorton[threadIdx.x / 8] = 0; // 默认是0
//	__syncthreads();*/
//
//	if (tid < nNodes)
//	{
//		uint16_t x, y, z;
//		if (tid < pactSize)
//		{
//			const int begSumNodes = d_sumNodesArray[blockIdx.x * blockDim.x];
//			/*if (threadIdx.x == 0) sh_nodeMorton[0] = d_sumNodesArray[blockIdx.x * blockDim.x];
//			__syncthreads();*/
//
//			const uint32_t key = d_pactDataArray[tid] & LOWER_3BIT_MASK;
//			const uint32_t morton = d_pactDataArray[tid] & D_MORTON_32_FLAG; // 去除符号位的实际莫顿码
//			// 得到mortonCode对应的实际存储节点的位置
//			const size_t address = d_sumNodesArray[tid] + key;
//
//			/*if (morton == 392)
//			{
//				printf("tid = %d, begSumNodes = %d, key = %d, d_pact = %d, morton = %d, d_sumNodesArray = %d, address = %d\n",
//					tid, (int)begSumNodes, (int)key, (int)(d_pactDataArray[tid] & D_MORTON_32_FLAG), (int)morton, (int)d_sumNodesArray[tid], (int)address);
//			}*/
//
//			d_nodeArray[address].mortonCode = morton;
//			morton3D_32_decode(morton, x, y, z);
//			d_nodeArray[address].origin = gridOrigin + width * Eigen::Vector3f((float)x, (float)y, (float)z);
//			d_nodeArray[address].width = width;
//
//			//d_morton2Idx[morton] = address; // 莫顿码到节点数组下标的映射
//
//			// (d_pactDataArray[tid] / 8) * 8 得到d_pactDataArray[tid](莫顿码)对应的以8为整数倍的下标
//			// 用于计算那些在这个if中没计算出来的节点莫顿码
//			//if ((morton / 8) * 8 != 0) sh_nodeMorton[threadIdx.x / 8] = (morton / 8) * 8; // 八个节点为一组
//			const int idx = (d_sumNodesArray[tid] - begSumNodes) / 8;
//			sh_nodeMorton[idx] = (morton / 8) * 8;
//
//			if (tid == 259)
//			{
//				printf("d_sumNodesArray[%d] = %d, begSumNodes = %d, morton = %d, sh_nodeMorton = \n",
//					tid, d_sumNodesArray[tid], begSumNodes, (int)morton, (int)(sh_nodeMorton[idx]));
//			}
//		}
//		//cg::sync(tile8);
//		__syncthreads();
//
//		/*if (tid == 159)
//		{
//			printf("sh_nodeMorton = %d\n", (int)sh_nodeMorton[32]);
//		}*/
//
//		// 计算不在voxel里的节点的莫顿码
//		if (d_nodeArray[tid].mortonCode == 0)
//		{
//			if (tid == 259)
//			{
//				printf("threadIdx.x = %d, sh_nodeMorton[%d] = %d\n", threadIdx.x, threadIdx.x / 8, (int)(sh_nodeMorton[threadIdx.x / 8]));
//			}
//
//			// threadIdx.x / 8不会越界，因为最坏情况也是连续7个CNode都是同一个父亲，也就是最多连续7个CNode的d_sumNodesArray是一样的
//			const uint32_t morton = (tid & LOWER_3BIT_MASK) + sh_nodeMorton[threadIdx.x / 8];
//			d_nodeArray[tid].mortonCode = morton;
//			morton3D_32_decode(morton, x, y, z);
//			d_nodeArray[tid].origin = gridOrigin + width * Eigen::Vector3f((float)x, (float)y, (float)z);
//			d_nodeArray[tid].width = width;
//
//			//d_morton2Idx[morton] = tid;
//		}
//	}
//}

//__global__ void createNode(const size_t nNodes,
//	const size_t pactSize,
//	const size_t d_preChildDepthTreeNodes, // 子节点层的前面所有层的节点数量(exclusive scan)，用于确定在总节点数组中的位置
//	const size_t d_preDepthTreeNodes, // 当前层的前面所有层的节点数量(exclusive scan)，用于确定在总节点数组中的位置
//	const size_t* d_sumNodesArray, // 这一层的节点数量inclusive scan数组
//	const uint32_t* d_pactDataArray,
//	const Eigen::Vector3f* d_gridOrigin,
//	const float* d_width,
//	SVONode* d_nodeArray,
//	SVONode* d_childArray/*,
//	size_t* d_morton2Idx*/)
//{
//	extern __shared__ uint32_t sh_nodeMorton[]; // blockSize / 8，数值为8的整数倍
//
//	/*cg::thread_block ctb = cg::this_thread_block();
//	cg::thread_group tile8 = cg::tiled_partition(ctb, 8);*/
//
//	size_t tid = threadIdx.x + blockIdx.x * blockDim.x;
//	const Eigen::Vector3f gridOrigin = *d_gridOrigin;
//	const float width = *d_width;
//	/*sh_nodeMorton[threadIdx.x / 8] = 0; // 默认是0
//	__syncthreads();*/
//
//	if (tid < nNodes)
//	{
//		uint16_t x, y, z;
//		if (tid < pactSize)
//		{
//			const int begSumNodes = d_sumNodesArray[blockIdx.x * blockDim.x];
//			/*const size_t preChildDepthTreeNodes = *d_preChildDepthTreeNodes;
//			const size_t preDepthTreeNodes = *d_preDepthTreeNodes;*/
//
//			const uint32_t key = d_pactDataArray[tid] & LOWER_3BIT_MASK;
//			const uint32_t morton = d_pactDataArray[tid] & D_MORTON_32_FLAG;
//			const size_t address = d_sumNodesArray[tid] + key;
//
//			SVONode& tNode = d_nodeArray[address];
//			tNode.mortonCode = morton;
//			morton3D_32_decode(morton, x, y, z);
//			tNode.origin = gridOrigin + width * Eigen::Vector3f((float)x, (float)y, (float)z);
//			tNode.width = width;
//			tNode.isLeaf = false;
//
//			/*if (tid == 0) printf("d_preChildDepthTreeNodes = %llu, d_preDepthTreeNodes = %llu\n", d_preChildDepthTreeNodes, d_preDepthTreeNodes);
//			if (tid == 0) printf("width = %f\n", width);*/
//
//#pragma unroll
//			for (int i = 0; i < 8; ++i)
//			{
//				tNode.childs[i] = d_preChildDepthTreeNodes + tid * 8 + i;
//				d_childArray[tid * 8 + i].parent = d_preDepthTreeNodes + tid;
//				//if (tid == 0 && i == 0) printf("child = %d, parent = %d\n", d_nodeArray[address].childs[i], d_childArray[tid * 8 + i].parent);
//			}
//
//			//d_morton2Idx[morton] = address; // 莫顿码到节点数组下标的映射
//
//			// (d_pactDataArray[tid] / 8) * 8 得到d_pactDataArray[tid](莫顿码)对应的以8为整数倍的下标
//			// 用于计算那些在这个if中没计算出来的节点莫顿码
//			//if ((morton / 8) * 8 != 0) sh_nodeMorton[threadIdx.x / 8] = (morton / 8) * 8; // 八个节点为一组
//			const int idx = (d_sumNodesArray[tid] - begSumNodes) / 8;
//			sh_nodeMorton[idx] = (morton / 8) * 8;
//		}
//		//cg::sync(tile8);
//		__syncthreads();
//
//		// 计算不在voxel里的节点的莫顿码(此时的节点为叶子节点)
//		SVONode& tNode = d_nodeArray[tid];
//		if (tNode.mortonCode == 0)
//		{
//			const uint32_t morton = (tid & LOWER_3BIT_MASK) + sh_nodeMorton[threadIdx.x / 8];
//
//			tNode.mortonCode = morton;
//			morton3D_32_decode(morton, x, y, z);
//			tNode.origin = gridOrigin + width * Eigen::Vector3f((float)x, (float)y, (float)z);
//			tNode.width = width;
//
//			//d_morton2Idx[morton] = tid;
//		}
//	}
//}

__global__ void createNode_1(const size_t pactSize,
	const size_t* d_sumNodesArray,
	const uint32_t* d_pactDataArray,
	const Eigen::Vector3f* d_gridOrigin,
	const float* d_width,
	uint32_t* d_begMortonArray,
	SVONode* d_nodeArray)
{
	size_t tid = threadIdx.x + blockIdx.x * blockDim.x;

	uint16_t x, y, z;
	if (tid < pactSize)
	{
		const Eigen::Vector3f gridOrigin = *d_gridOrigin;
		const float width = *d_width;

		const int sumNodes = d_sumNodesArray[tid];
		const uint32_t pactData = d_pactDataArray[tid];

		const uint32_t key = pactData & LOWER_3BIT_MASK;
		const uint32_t morton = pactData & D_MORTON_32_FLAG; // 去除符号位的实际莫顿码
		// 得到mortonCode对应的实际存储节点的位置
		const size_t address = sumNodes + key;

		SVONode& tNode = d_nodeArray[address];
		tNode.mortonCode = morton;
		morton3D_32_decode(morton, x, y, z);
		tNode.origin = gridOrigin + width * Eigen::Vector3f((float)x, (float)y, (float)z);
		tNode.width = width;

		d_begMortonArray[tid] = (morton / 8) * 8;
	}
}

__global__ void createNode_2(const size_t pactSize,
	const size_t d_preChildDepthTreeNodes, // 子节点层的前面所有层的节点数量(exclusive scan)，用于确定在总节点数组中的位置
	const size_t d_preDepthTreeNodes, // 当前层的前面所有层的节点数量(exclusive scan)，用于确定在总节点数组中的位置
	const size_t* d_sumNodesArray, // 这一层的节点数量inclusive scan数组
	const uint32_t* d_pactDataArray,
	const Eigen::Vector3f* d_gridOrigin,
	const float* d_width,
	uint32_t* d_begMortonArray,
	SVONode* d_nodeArray,
	SVONode* d_childArray)
{
	size_t tid = threadIdx.x + blockIdx.x * blockDim.x;

	uint16_t x, y, z;
	if (tid < pactSize)
	{
		const Eigen::Vector3f gridOrigin = *d_gridOrigin;
		const float width = *d_width;

		const int sumNodes = d_sumNodesArray[tid];
		const uint32_t pactData = d_pactDataArray[tid];

		const uint32_t key = pactData & LOWER_3BIT_MASK;
		const uint32_t morton = pactData & D_MORTON_32_FLAG;
		const size_t address = sumNodes + key;

		SVONode& tNode = d_nodeArray[address];
		tNode.mortonCode = morton;
		morton3D_32_decode(morton, x, y, z);
		tNode.origin = gridOrigin + width * Eigen::Vector3f((float)x, (float)y, (float)z);
		tNode.width = width;
		tNode.isLeaf = false;

		//printf("tid = %llu, morton = %d\n", tid, (int)(morton));
		d_begMortonArray[tid] = (morton / 8) * 8;

#pragma unroll
		for (int i = 0; i < 8; ++i)
		{
			tNode.childs[i] = d_preChildDepthTreeNodes + tid * 8 + i;
			d_childArray[tid * 8 + i].parent = d_preDepthTreeNodes + sumNodes + key; // +key是因为后面还需要构建remain nodes
		}
	}
}

__global__ void createRemainNode(const size_t nNodes,
	const Eigen::Vector3f* d_gridOrigin,
	const float* d_width,
	const uint32_t* d_begMortonArray,
	SVONode* d_nodeArray)
{
	extern __shared__ uint32_t sh_begMortonArray[];
	size_t tid = threadIdx.x + blockIdx.x * blockDim.x;

	uint16_t x, y, z;
	if (tid < nNodes)
	{
		sh_begMortonArray[threadIdx.x / 8] = d_begMortonArray[tid / 8];
		/*printf("threadIdx.x = %d, tid = %llu, d_begMortonArray[%llu] = %d, sh_begMortonArray[%d] = %d\n",
			threadIdx.x, tid, tid / 8, (int)d_begMortonArray[tid / 8], threadIdx.x / 8, (int)sh_begMortonArray[threadIdx.x / 8]);*/
		__syncthreads();

		if (d_nodeArray[tid].mortonCode == 0)
		{
			const Eigen::Vector3f gridOrigin = *d_gridOrigin;
			const float width = *d_width;

			const uint32_t key = tid & LOWER_3BIT_MASK;
			const uint32_t morton = sh_begMortonArray[threadIdx.x / 8] + key;

			SVONode& tNode = d_nodeArray[tid];
			tNode.mortonCode = morton;
			//printf("tid = %llu, morton = %d\n", tid, (int)(d_nodeArray[tid].mortonCode));

			morton3D_32_decode(morton, x, y, z);
			tNode.origin = gridOrigin + width * Eigen::Vector3f((float)x, (float)y, (float)z);
			tNode.width = width;
		}
	}
}

void SparseVoxelOctree::createOctree()
{
	assert(surfaceVoxelGridSize.x() >= 1 && surfaceVoxelGridSize.y() >= 1 && surfaceVoxelGridSize.z() >= 1);
	size_t gridCNodeSize = (size_t)mortonEncode_LUT((uint16_t)(surfaceVoxelGridSize.x() - 1), (uint16_t)(surfaceVoxelGridSize.y() - 1), (uint16_t)(surfaceVoxelGridSize.z() - 1)) + 1;
	//size_t gridCNodeSize = (size_t)((size_t)surfaceVoxelGridSize.x() * (size_t)surfaceVoxelGridSize.y() * (size_t)surfaceVoxelGridSize.z());
	size_t gridTreeNodeSize = gridCNodeSize % 8 ? gridCNodeSize + 8 - (gridCNodeSize % 8) : gridCNodeSize;
	///	TODO: 调成同样大小(只要把模型的bbox设置为立方体就可以了，具体可参考cuda_voxelizer中的createMeshBBCube方法)
	Eigen::Vector3f unitVoxelSize = Eigen::Vector3f(modelBBox.width.x() / surfaceVoxelGridSize.x(),
		modelBBox.width.y() / surfaceVoxelGridSize.y(),
		modelBBox.width.z() / surfaceVoxelGridSize.z());
	float unitNodeWidth = unitVoxelSize.x();

	Eigen::Vector3i* d_surfaceVoxelGridSize;
	CUDA_CHECK(hipMalloc((void**)&d_surfaceVoxelGridSize, sizeof(Eigen::Vector3i)));
	CUDA_CHECK(hipMemcpy(d_surfaceVoxelGridSize, &surfaceVoxelGridSize, sizeof(Eigen::Vector3i), hipMemcpyHostToDevice));
	Eigen::Vector3f* d_gridOrigin;
	CUDA_CHECK(hipMalloc((void**)&d_gridOrigin, sizeof(Eigen::Vector3f)));
	CUDA_CHECK(hipMemcpy(d_gridOrigin, &modelBBox.min, sizeof(Eigen::Vector3f), hipMemcpyHostToDevice));
	Eigen::Vector3f* d_unitVoxelSize;
	CUDA_CHECK(hipMalloc((void**)&d_unitVoxelSize, sizeof(Eigen::Vector3f)));
	CUDA_CHECK(hipMemcpy(d_unitVoxelSize, &unitVoxelSize, sizeof(Eigen::Vector3f), hipMemcpyHostToDevice));
	float* d_unitNodeWidth;
	CUDA_CHECK(hipMalloc((void**)&d_unitNodeWidth, sizeof(float)));
	CUDA_CHECK(hipMemcpy(d_unitNodeWidth, &unitNodeWidth, sizeof(float), hipMemcpyHostToDevice));

	/*CUDA_CHECK(hipMemcpyToSymbol(HIP_SYMBOL(d_surfaceVoxelGridSize), &surfaceVoxelGridSize, sizeof(Eigen::Vector3i)));
	CUDA_CHECK(hipMemcpyToSymbol(HIP_SYMBOL(d_gridOrigin), &modelBBox.min, sizeof(Eigen::Vector3f)));
	CUDA_CHECK(hipMemcpyToSymbol(HIP_SYMBOL(d_unitVoxelSize), &unitVoxelSize, sizeof(Eigen::Vector3f)));
	CUDA_CHECK(hipMemcpyToSymbol(HIP_SYMBOL(d_unitNodeWidth), &unitNodeWidth, sizeof(float)));*/

	thrust::device_vector<uint32_t> d_CNodeMortonArray(gridCNodeSize, 0);
	thrust::device_vector<bool> d_isValidCNodeArray;
	thrust::device_vector<size_t> d_esumCNodesArray; // exclusive scan
	thrust::device_vector<uint32_t> d_pactCNodeArray;
	thrust::device_vector<size_t> d_numTreeNodesArray; // 节点数量记录数组
	thrust::device_vector<size_t> d_sumTreeNodesArray; // inlusive scan
	thrust::device_vector<size_t> d_esumTreeNodesArray; // 存储每一层节点数量的exclusive scan数组
	thrust::device_vector<uint32_t> d_begMortonArray;
	thrust::device_vector<SVONode> d_nodeArray; // 存储某一层的节点数组
	thrust::device_vector<SVONode> d_SVONodeArray; // save all sparse octree nodes

	// mesh voxelize
	//size_t voxelArraySize = (size_t)((size_t)(surfaceVoxelGridSize.x() + 1) * (size_t)(surfaceVoxelGridSize.y() + 1) * (size_t)(surfaceVoxelGridSize.z() + 1));
	// 不需要+1（莫顿码为0代表坐标位于原点的第一个八叉树节点，八个顶点坐标需要令算）
	resizeThrust(d_CNodeMortonArray, gridCNodeSize, (uint32_t)0);
	meshVoxelize(d_surfaceVoxelGridSize, d_unitVoxelSize, d_gridOrigin, d_CNodeMortonArray);

	//#ifndef NDEBUG
	//	vector<uint32_t> h_CNodeMortonArray(gridCNodeSize, 0);
	//	int cnodes = 0;
	//	CUDA_CHECK(hipMemcpy(h_CNodeMortonArray.data(), d_CNodeMortonArray.data().get(), sizeof(uint32_t) * gridCNodeSize, hipMemcpyDeviceToHost));
	//	for (int i = 0; i < h_CNodeMortonArray.size(); ++i)
	//		if (h_CNodeMortonArray[i] != 0) { ++cnodes; std::cout << (h_CNodeMortonArray[i] & D_MORTON_32_FLAG) << std::endl; }
	//	std::cout << "cnodes = " << cnodes << "\n--------\n";
	//#endif // !NDEBUG

		// create octree
		// 最后出来的树会比原始模型大7个格子, TODO: 到最顶层的时候只建立与模型bb相同的一个格子，它的周围7个格子不要建出来
	while (true)
	{
		// compute the number of 'coarse nodes'(eg: voxels)
		//size_t pactCNodeArraySize = 0;
		resizeThrust(d_isValidCNodeArray, gridCNodeSize);
		resizeThrust(d_esumCNodesArray, gridCNodeSize);
		thrust::transform(d_CNodeMortonArray.begin(), d_CNodeMortonArray.end(), d_isValidCNodeArray.begin(), scanMortonFlag<uint32_t>());
		thrust::exclusive_scan(d_isValidCNodeArray.begin(), d_isValidCNodeArray.end(), d_esumCNodesArray.begin(), 0); // 必须加init
		/*for (int i = 0; i < d_esumCNodesArray.size(); ++i)
			if (d_esumCNodesArray[i] != 0) std::cout << d_esumCNodesArray[i] << std::endl;*/
		size_t numCNodes = *(d_esumCNodesArray.rbegin()) + *(d_isValidCNodeArray.rbegin());
		//std::cout << "numCNodes = " << numCNodes << "\n--------\n";
		if (!numCNodes) { printf("Sparse Voxel Octree depth: %d\n", treeDepth); break; }

		treeDepth++;

		// compact coarse node array
		d_pactCNodeArray.clear(); resizeThrust(d_pactCNodeArray, numCNodes);
		getOccupancyMaxPotentialBlockSize(gridCNodeSize, minGridSize, blockSize, gridSize, compactArray, 0, 0);
		compactArray << <gridSize, blockSize >> > (gridCNodeSize, d_isValidCNodeArray.data().get(),
			d_CNodeMortonArray.data().get(), d_esumCNodesArray.data().get(), d_pactCNodeArray.data().get());
		getLastCudaError("Kernel 'compactArray' launch failed!\n");
		vector<uint32_t> h_pactCNodeArray(numCNodes, 0);
		CUDA_CHECK(hipMemcpy(h_pactCNodeArray.data(), d_pactCNodeArray.data().get(), sizeof(uint32_t) * numCNodes, hipMemcpyDeviceToHost));
		/*for (int i = 0; i < h_pactCNodeArray.size(); ++i)
			std::cout << (h_pactCNodeArray[i] & D_MORTON_32_FLAG) << std::endl;
		std::cout << "--------\n";*/

		// 验证体素
#ifdef NDEBUG
		if (treeDepth == 1)
		{
			vector<uint32_t> voxelArray;
			voxelArray.resize(numCNodes);
			CUDA_CHECK(hipMemcpy(voxelArray.data(), d_pactCNodeArray.data().get(), sizeof(uint32_t) * numCNodes, hipMemcpyDeviceToHost));
			writeVoxel(voxelArray, "bunny", unitNodeWidth);
		}
#endif // !NDEBUG

		// compute the number of (real)octree nodes by coarse node array
		// and set parent's morton code to 'd_CNodeMortonArray'
		size_t numNodes = 1;
		if (numCNodes > 1)
		{
			resizeThrust(d_numTreeNodesArray, numCNodes, (size_t)0);
			d_CNodeMortonArray.clear(); resizeThrust(d_CNodeMortonArray, gridTreeNodeSize, (uint32_t)0); // 此时用于记录父节点层的coarse node
			getOccupancyMaxPotentialBlockSize(numCNodes, minGridSize, blockSize, gridSize, cpNumNodes, 0, 0);
			const uint32_t firstMortonCode = getParentMorton(d_pactCNodeArray[0]);
			d_CNodeMortonArray[firstMortonCode] = firstMortonCode | E_MORTON_32_FLAG;
			cpNumNodes << <gridSize, blockSize >> > (numCNodes, d_pactCNodeArray.data().get(), d_numTreeNodesArray.data().get(), d_CNodeMortonArray.data().get());
			getLastCudaError("Kernel 'cpNumNodes' launch failed!\n");
			resizeThrust(d_sumTreeNodesArray, numCNodes, (size_t)0); // inlusive scan
			thrust::inclusive_scan(d_numTreeNodesArray.begin(), d_numTreeNodesArray.end(), d_sumTreeNodesArray.begin());

			/*vector<uint32_t> h_CNodeMortonArray(numCNodes, 0);
			CUDA_CHECK(hipMemcpy(h_CNodeMortonArray.data(), d_CNodeMortonArray.data().get(), sizeof(uint32_t) * numCNodes, hipMemcpyDeviceToHost));
			for (int i = 0; i < h_CNodeMortonArray.size(); ++i)
				if (h_CNodeMortonArray[i] != 0) std::cout << "parent: " << (h_CNodeMortonArray[i] & D_MORTON_32_FLAG) << std::endl;
			std::cout << "--------\n";*/

			/*std::cout << "current coarse node morton:\n";
			vector<short> h_numTreeNodesArray(numCNodes, 0);
			vector<size_t> h_sumTreeNodesArray(numCNodes, 0);
			CUDA_CHECK(hipMemcpy(h_numTreeNodesArray.data(), d_numTreeNodesArray.data().get(), sizeof(short) * numCNodes, hipMemcpyDeviceToHost));
			CUDA_CHECK(hipMemcpy(h_sumTreeNodesArray.data(), d_sumTreeNodesArray.data().get(), sizeof(size_t) * numCNodes, hipMemcpyDeviceToHost));
			for (int i = 0; i < h_numTreeNodesArray.size(); ++i)
				if (i == 0)
					std::cout << (h_pactCNodeArray[i] & D_MORTON_32_FLAG) << ", " << h_numTreeNodesArray[i] << ", " << h_sumTreeNodesArray[i] << std::endl;
				else
					std::cout << (h_pactCNodeArray[i] & D_MORTON_32_FLAG) << ", " << (h_pactCNodeArray[i - 1] & D_MORTON_32_FLAG)
					<< ", " << h_numTreeNodesArray[i] << ", " << h_sumTreeNodesArray[i] << std::endl;
			std::cout << "--------\n";*/

			numNodes = *(d_sumTreeNodesArray.rbegin()) + 8;
			depthNumNodes.emplace_back(numNodes);
		}

		// set octree node array
		d_nodeArray.clear(); resizeThrust(d_nodeArray, numNodes, SVONode());
		uint32_t maxMortonCode = (*d_pactCNodeArray.rbegin()) & D_MORTON_32_FLAG;
		//resizeThrust(d_morton2Idx, gridTreeNodeSize);
		d_begMortonArray.clear(); resizeThrust(d_begMortonArray, numCNodes);
		if (treeDepth < 2)
		{
			//createNode << <gridSize, blockSize, sizeof(uint32_t)* blockSize >> > (numNodes, numCNodes, d_sumTreeNodesArray.data().get(),
			//	d_pactCNodeArray.data().get(), d_gridOrigin, d_unitNodeWidth, d_nodeArray.data().get()/*, d_morton2Idx.data().get()*/);

			getOccupancyMaxPotentialBlockSize(numCNodes, minGridSize, blockSize, gridSize, createNode_1);
			createNode_1 << <gridSize, blockSize >> > (numCNodes, d_sumTreeNodesArray.data().get(),
				d_pactCNodeArray.data().get(), d_gridOrigin, d_unitNodeWidth, d_begMortonArray.data().get(), d_nodeArray.data().get());
			getLastCudaError("Kernel 'createNode_1' launch failed!\n");
			printf("444\n");

			/*vector<SVONode> h_nodeArray(numNodes);
			CUDA_CHECK(hipMemcpy(h_nodeArray.data(), d_nodeArray.data().get(), sizeof(SVONode) * numNodes, hipMemcpyDeviceToHost));
			for (int i = 0; i < h_nodeArray.size(); ++i)
				std::cout << "node morton: " << (h_nodeArray[i].mortonCode) << std::endl;
			std::cout << "--------\n";*/

			d_esumTreeNodesArray.push_back(0);
		}
		else
		{
			//createNode << <gridSize, blockSize, sizeof(uint32_t)* blockSize >> > (numNodes, numCNodes,
			//	*(d_esumTreeNodesArray.rbegin() + 1), *(d_esumTreeNodesArray.rbegin()),
			//	d_sumTreeNodesArray.data().get(), d_pactCNodeArray.data().get(), d_gridOrigin, d_unitNodeWidth, d_nodeArray.data().get(),
			//	(d_SVONodeArray.data() + (*(d_esumTreeNodesArray.rbegin() + 1))).get()/*, d_morton2Idx.data().get()*/);

			getOccupancyMaxPotentialBlockSize(numCNodes, minGridSize, blockSize, gridSize, createNode_2);
			createNode_2 << <gridSize, blockSize >> > (numCNodes, *(d_esumTreeNodesArray.rbegin() + 1), *(d_esumTreeNodesArray.rbegin()),
				d_sumTreeNodesArray.data().get(), d_pactCNodeArray.data().get(), d_gridOrigin, d_unitNodeWidth, d_begMortonArray.data().get(),
				d_nodeArray.data().get(), (d_SVONodeArray.data() + (*(d_esumTreeNodesArray.rbegin() + 1))).get());
			getLastCudaError("Kernel 'createNode_2' launch failed!\n");

			/*vector<SVONode> h_nodeArray(numNodes);
			CUDA_CHECK(hipMemcpy(h_nodeArray.data(), d_nodeArray.data().get(), sizeof(SVONode) * numNodes, hipMemcpyDeviceToHost));
			for (int i = 0; i < h_nodeArray.size(); ++i)
				std::cout << "node morton: " << (h_nodeArray[i].mortonCode) << std::endl;
			std::cout << "--------\n";*/
		}
		auto newEndOfBegMorton = thrust::unique(d_begMortonArray.begin(), d_begMortonArray.end());
		resizeThrust(d_begMortonArray, newEndOfBegMorton - d_begMortonArray.begin());

		/*for (int i = 0; i < d_begMortonArray.size(); ++i)
			std::cout << "begMorton: " << d_begMortonArray[i] << std::endl;
		std::cout << "--------\n";*/

		blockSize = 256; gridSize = (numNodes + blockSize - 1) / blockSize;
		createRemainNode << <gridSize, blockSize, sizeof(uint32_t)* blockSize / 8 >> > (numNodes, d_gridOrigin, d_unitNodeWidth,
			d_begMortonArray.data().get(), d_nodeArray.data().get());
		getLastCudaError("Kernel 'createRemainNode' launch failed!\n");

		/*vector<SVONode> h_nodeArray(numNodes);
		CUDA_CHECK(hipMemcpy(h_nodeArray.data(), d_nodeArray.data().get(), sizeof(SVONode) * numNodes, hipMemcpyDeviceToHost));
		for (int i = 0; i < h_nodeArray.size(); ++i)
			std::cout << "all node morton: " << (h_nodeArray[i].mortonCode) << std::endl;
		std::cout << "--------\n";*/

		d_SVONodeArray.insert(d_SVONodeArray.end(), d_nodeArray.begin(), d_nodeArray.end());
		//d_allSVONodeArray.push_back(d_nodeArray);
		//d_allMorton2Idx.push_back(d_morton2Idx);

		/*if (treeDepth >= 2)
		{
			vector<SVONode> h_SVONodeArray(d_SVONodeArray.size());
			CUDA_CHECK(hipMemcpy(h_SVONodeArray.data(), d_SVONodeArray.data().get(), sizeof(SVONode) * d_SVONodeArray.size(), hipMemcpyDeviceToHost));
			for (int i = 0; i < h_SVONodeArray.size(); ++i)
			{
				std::cout << (h_SVONodeArray[i].parent) << " -- ";
				for (int j = 0; j < 8; ++j)
					std::cout << (h_SVONodeArray[i].childs[j]) << ", ";
				std::cout << '\n';
			}
			std::cout << "--------\n";
		}*/

		d_esumTreeNodesArray.push_back(numNodes + (*d_esumTreeNodesArray.rbegin()));

		/*for (int i = 0; i < d_esumTreeNodesArray.size(); ++i)
			std::cout << d_esumTreeNodesArray[i] << std::endl;
		std::cout << "--------\n";*/

		//// special condition
		//if (treeDepth == 1 && gridCNodeSize == 1) { printf("Sparse Voxel Octree depth: %d\n", treeDepth); break; }
		// resize parent array 'd_CNodeMortonArray' to nexe loop
		uint32_t numParentCNodes = *thrust::max_element(d_CNodeMortonArray.begin(), d_CNodeMortonArray.end());
		bool isValidMorton = (numParentCNodes >> 31) & 1;
		// '+ isValidMorton' to prevent '(numParentNodes & D_MORTON_32_FLAG) = 0'同时正好可以让最后的大小能存储到最大的莫顿码
		numParentCNodes = (numParentCNodes & D_MORTON_32_FLAG) + isValidMorton;
		if (numParentCNodes != 0)
		{
			resizeThrust(d_CNodeMortonArray, numParentCNodes);
			unitNodeWidth *= 2.0; CUDA_CHECK(hipMemcpy(d_unitNodeWidth, &unitNodeWidth, sizeof(float), hipMemcpyHostToDevice));
			gridCNodeSize = numParentCNodes; gridTreeNodeSize = gridCNodeSize % 8 ? gridCNodeSize + 8 - (gridCNodeSize % 8) : gridCNodeSize;
			if (numNodes / 8 == 0) { printf("Sparse Voxel Octree depth: %d\n", treeDepth); break; }
		}
		else { printf("Sparse Voxel Octree depth: %d\n", treeDepth); break; }
	}
	numTreeNodes = d_esumTreeNodesArray[treeDepth];
	//TODO: copy to host
	svoNodeArray.resize(numTreeNodes);
	auto freeResOfCreateTree = [&]()
	{
		cleanupThrust(d_CNodeMortonArray);
		cleanupThrust(d_isValidCNodeArray);
		cleanupThrust(d_esumCNodesArray);
		cleanupThrust(d_pactCNodeArray);
		cleanupThrust(d_numTreeNodesArray);
		cleanupThrust(d_sumTreeNodesArray);
		cleanupThrust(d_nodeArray);

		CUDA_CHECK(hipFree(d_surfaceVoxelGridSize));
		CUDA_CHECK(hipFree(d_gridOrigin));
		CUDA_CHECK(hipFree(d_unitNodeWidth));
		CUDA_CHECK(hipFree(d_unitVoxelSize));
	};
	freeResOfCreateTree();

	constructNodeAtrributes(d_esumTreeNodesArray, d_SVONodeArray);
	CUDA_CHECK(hipMemcpy(svoNodeArray.data(), d_SVONodeArray.data().get(), sizeof(SVONode) * numTreeNodes, hipMemcpyDeviceToHost));
	cleanupThrust(d_numTreeNodesArray);
	cleanupThrust(d_SVONodeArray);
}

__device__ size_t d_topNodeIdx;
template<bool topFlag>
__global__ void findNeighbors(const size_t nNodes,
	const size_t preESumTreeNodes,
	SVONode* d_nodeArray)
{
	if (topFlag)
	{
		d_nodeArray[0].neighbors[13] = d_topNodeIdx;
	}
	else
	{
		size_t tid_x = threadIdx.x + blockIdx.x * blockDim.x;
		size_t tid_y = threadIdx.y + blockIdx.y * blockDim.y;

		if (tid_x < nNodes && tid_y < 27)
		{
			SVONode t = d_nodeArray[preESumTreeNodes + tid_x];
			SVONode p = d_nodeArray[t.parent];
			const uint8_t key = (t.mortonCode) & LOWER_3BIT_MASK;
			const unsigned int p_neighborIdx = p.neighbors[neighbor_LUTparent[key][tid_y]];
			if (p_neighborIdx != UINT32_MAX)
			{
				SVONode h = d_nodeArray[p_neighborIdx];
				t.neighbors[tid_y] = h.childs[neighbor_LUTchild[key][tid_y]];
			}
			else t.neighbors[tid_y] = UINT32_MAX;
		}
	}

}

void SparseVoxelOctree::constructNodeNeighbors(const thrust::device_vector<size_t>& d_esumTreeNodesArray,
	thrust::device_vector<SVONode>& d_SVONodeArray)
{
	dim3 gridSize, blockSize;
	blockSize.x = 32, blockSize.y = 32;
	gridSize.y = 1;
	// find neighbors(up to bottom)
	if (treeDepth >= 2)
	{
		const size_t idx = d_SVONodeArray.size() - 1;
		CUDA_CHECK(hipMemcpyToSymbol(HIP_SYMBOL(d_topNodeIdx), &idx, sizeof(size_t)));
		findNeighbors<true> << <1, 1 >> > (1, 0, (d_SVONodeArray.data() + idx).get());
		for (int i = treeDepth - 2; i >= 0; --i)
		{
			const size_t nNodes = depthNumNodes[i];
			gridSize.x = (nNodes + blockSize.x - 1) / blockSize.x;
			findNeighbors<false> << <gridSize, blockSize >> > (nNodes, d_esumTreeNodesArray[i], d_SVONodeArray.data().get());
		}
	}
}

//template<typename T1, typename T2>
//struct cuPair
//{
//	T1 first;
//	T2 second;
//	CUDA_CALLABLE_MEMBER NodeVertexPair(const T1& _first, const T2& _second) :first(_first), second(_second) {}
//};
__constant__ short int d_vertSharedLUT[64] =
{
	0, 1, 3, 4, 9, 10, 12, 13,

	1, 2, 4, 5, 10, 11, 13 ,14,

	3, 4, 6, 7, 12, 13, 15, 16,

	4, 5, 7, 8, 13, 14, 16, 17,

	9, 10, 12, 13, 18, 19, 21, 22,

	10, 11, 13, 14, 19, 20, 22, 23,

	12, 13, 15, 16, 21, 22, 24, 25,

	13, 14, 16, 17, 22, 23, 25, 26
};
__global__ void determineNodeVertex(const size_t nNodes,
	const SVONode* d_nodeArray,
	thrust::pair<Eigen::Vector3f, uint32_t>* d_nodeVertArray)
{
	size_t tid_x = threadIdx.x + blockIdx.x * blockDim.x;

	if (tid_x < nNodes)
	{
		uint16_t x, y, z;
		Eigen::Vector3f origin = d_nodeArray[tid_x].origin;
		float width = d_nodeArray[tid_x].width;
#pragma unroll
		for (int i = 0; i < 8; ++i)
		{
			morton3D_32_decode(i, x, y, z);
			Eigen::Vector3f corner = width * Eigen::Vector3f((float)x, (float)y, (float)z);

			uint32_t morton = UINT_MAX, idx = tid_x;
			for (int j = 0; j < 8; ++j)
				if (d_nodeArray[tid_x].neighbors[d_vertSharedLUT[i * 8 + j]] < morton) idx = d_nodeArray[tid_x].neighbors[d_vertSharedLUT[i * 8 + j]];

			d_nodeVertArray[tid_x * 8 + i] = thrust::make_pair(corner, idx);
		}
	}
}

// edge: 02 23 31 10   46 67 75 54   04 26 37 15 
__constant__ short int d_edgeSharedLUT[48] =
{
	 3, 4, 12, 13,
	 4, 7, 13, 16,
	 4, 5, 13, 14,
	 1, 4, 10, 13,

	 12, 13, 21, 22,
	 13, 16, 22, 25,
	 13, 14, 22, 23,
	 10, 13, 19, 22,

	 9, 10, 12, 13,
	 12, 13, 15, 16,
	 13, 14, 16, 17,
	 10, 11, 13, 14
};
__global__ void determineNodeEdge(const size_t nNodes,
	const SVONode* d_nodeArray,
	thrust::pair<thrust_edge, uint32_t>* d_nodeEdgeArray)
{
	size_t tid_x = threadIdx.x + blockIdx.x * blockDim.x;

	if (tid_x < nNodes)
	{
		Eigen::Vector3f origin = d_nodeArray[tid_x].origin;
		float width = d_nodeArray[tid_x].width;

		thrust_edge edges[12] =
		{
			thrust::make_pair(origin, origin + Eigen::Vector3f(0, width, 0)),
			thrust::make_pair(origin + Eigen::Vector3f(0, width, 0), origin + Eigen::Vector3f(width, width, 0)),
			thrust::make_pair(origin + Eigen::Vector3f(width, width, 0), origin + Eigen::Vector3f(width, 0, 0)),
			thrust::make_pair(origin + Eigen::Vector3f(width, 0, 0), origin),

			thrust::make_pair(origin + Eigen::Vector3f(0, 0, width), origin + Eigen::Vector3f(0, width, width)),
			thrust::make_pair(origin + Eigen::Vector3f(0, width, width), origin + Eigen::Vector3f(width, width, width)),
			thrust::make_pair(origin + Eigen::Vector3f(width, width, width), origin + Eigen::Vector3f(width, 0, width)),
			thrust::make_pair(origin + Eigen::Vector3f(width, 0, width), origin + Eigen::Vector3f(0, 0, width)),

			thrust::make_pair(origin, origin + Eigen::Vector3f(0, 0, width)),
			thrust::make_pair(origin + Eigen::Vector3f(0, width, 0), origin + Eigen::Vector3f(0, width, width)),
			thrust::make_pair(origin + Eigen::Vector3f(width, width, 0), origin + Eigen::Vector3f(width, width, width)),
			thrust::make_pair(origin + Eigen::Vector3f(width, 0, 0), origin + Eigen::Vector3f(width, 0, width)),
		};

#pragma unroll
		for (int i = 0; i < 12; ++i)
		{
			thrust_edge edge = edges[i];

			uint32_t morton = UINT_MAX, idx = tid_x;
			for (int j = 0; j < 4; ++j)
				if (d_nodeArray[tid_x].neighbors[d_edgeSharedLUT[i * 4 + j]] < morton) idx = d_nodeArray[tid_x].neighbors[d_edgeSharedLUT[i * 4 + j]];

			d_nodeEdgeArray[tid_x * 12 + i] = thrust::make_pair(edge, idx);
		}
	}
}

template <typename T>
struct uniqueVert : public thrust::binary_function<T, T, T> {
	__host__ __device__ bool operator()(const T& a, const T& b) {
		return a.first == b.first;
	}
};

template <typename T>
struct uniqueEdge : public thrust::binary_function<T, T, T> {
	__host__ __device__
		bool operator()(const T& a, const T& b) {
		return ((a.first.first == b.first.first) && (a.first.second == b.first.second))||
				((a.first.first == b.first.second) && (a.first.second == b.first.first));
	}
};

void SparseVoxelOctree::constructNodeVertexAndEdge(thrust::device_vector<SVONode>& d_SVONodeArray)
{
	hipStream_t streams[2];
	for (int i = 0; i < 2; ++i) CUDA_CHECK(hipStreamCreate(&streams[i]));

	thrust::device_vector < thrust::pair<Eigen::Vector3f, uint32_t>> d_nodeVertArray(numTreeNodes * 8);
	getOccupancyMaxPotentialBlockSize(numTreeNodes, minGridSize, blockSize, gridSize, determineNodeVertex, 0, 0);
	determineNodeVertex << <gridSize, blockSize, 0, streams[0] >> > (numTreeNodes, d_SVONodeArray.data().get(), d_nodeVertArray.data().get());

	thrust::device_vector < thrust::pair<thrust_edge, uint32_t>> d_nodeEdgeArray(numTreeNodes * 12);
	getOccupancyMaxPotentialBlockSize(numTreeNodes, minGridSize, blockSize, gridSize, determineNodeEdge, 0, 0);
	determineNodeEdge << <gridSize, blockSize, 0, streams[1] >> > (numTreeNodes, d_SVONodeArray.data().get(), d_nodeEdgeArray.data().get());

	hipStreamSynchronize(streams[0]);
	auto vertNewEnd = thrust::unique(d_nodeVertArray.begin(), d_nodeVertArray.end(), uniqueVert<thrust::pair<Eigen::Vector3f, uint32_t>>());
	const size_t numVerts = vertNewEnd - d_nodeVertArray.begin();
	resizeThrust(d_nodeVertArray, numVerts);
	nodeVertexArray.resize(numVerts);
	CUDA_CHECK(hipMemcpy(nodeVertexArray.data(), d_nodeVertArray.data().get(),
		sizeof(thrust::pair<Eigen::Vector3f, uint32_t>) * numVerts, hipMemcpyDeviceToHost));

	hipStreamSynchronize(streams[1]);
	auto edgeNewEnd = thrust::unique(d_nodeEdgeArray.begin(), d_nodeEdgeArray.end(), uniqueEdge<thrust::pair<thrust_edge, uint32_t>>()); // error
	const size_t numEdges = edgeNewEnd - d_nodeEdgeArray.begin();
	resizeThrust(d_nodeEdgeArray, numEdges);
	nodeEdgeArray.resize(numEdges);
	CUDA_CHECK(hipMemcpy(nodeEdgeArray.data(), d_nodeEdgeArray.data().get(),
		sizeof(thrust::pair<thrust_edge, uint32_t>) * numEdges, hipMemcpyDeviceToHost));

	for (int i = 0; i < 2; ++i) CUDA_CHECK(hipStreamDestroy(streams[i]));
}

void SparseVoxelOctree::constructNodeAtrributes(const thrust::device_vector<size_t>& d_esumTreeNodesArray,
	thrust::device_vector<SVONode>& d_SVONodeArray)
{
	constructNodeNeighbors(d_esumTreeNodesArray, d_SVONodeArray);

	constructNodeVertexAndEdge(d_SVONodeArray);
	//constructNodeEdgeArray();
}

void SparseVoxelOctree::writeTree(const std::string base_filename)
{
	std::string filename_output = base_filename + std::string("_") + std::to_string(treeDepth) + std::string("_tree.obj");
	std::ofstream output(filename_output.c_str(), std::ios::out);
	assert(output);

#ifndef SILENT
	fprintf(stdout, "[I/O] Writing octree data in obj format to file %s \n", filename_output.c_str());
	// Write stats
	size_t voxels_seen = 0;
	const size_t write_stats_25 = numTreeNodes / 4.0f;
	fprintf(stdout, "[I/O] Writing to file: 0%%...");
#endif

	size_t faceBegIdx = 0;
	for (const auto& node : svoNodeArray)
	{
#ifndef SILENT			
		voxels_seen++;
		if (voxels_seen == write_stats_25) { fprintf(stdout, "25%%..."); }
		else if (voxels_seen == write_stats_25 * size_t(2)) { fprintf(stdout, "50%%..."); }
		else if (voxels_seen == write_stats_25 * size_t(3)) { fprintf(stdout, "75%%..."); }
#endif
		/*if (faceBegIdx == 440)
		{
			std::cout << node.origin.transpose() << std::endl;
		}*/
		//std::cout << "node.origin = " << node.origin.transpose() << ", node.width = " << node.width << ", node.morton = " << node.mortonCode << '\n';

		write_cube(node.origin, Eigen::Vector3f(node.width, node.width, node.width), output, faceBegIdx);
	}
#ifndef SILENT
	fprintf(stdout, "100%% \n");
#endif

	output.close();
}

void SparseVoxelOctree::writeVoxel(const vector<uint32_t>& voxelArray, const std::string& base_filename, const float& width)
{
	std::string filename_output = base_filename + std::string("_") + std::to_string(treeDepth) + std::string("_voxel.obj");
	std::ofstream output(filename_output.c_str(), std::ios::out);
	assert(output);

#ifndef SILENT
	fprintf(stdout, "[I/O] Writing data in obj voxels format to file %s \n", filename_output.c_str());
	// Write stats
	/*size_t voxels_seen = 0;
	const size_t write_stats_25 = numTreeNodes / 4.0f;*/
	//fprintf(stdout, "[I/O] Writing to file: 0%%...");
#endif

	size_t faceBegIdx = 0;
	for (size_t i = 0; i < voxelArray.size(); ++i)
	{
		//#ifndef SILENT			
		//		voxels_seen++;
		//		if (voxels_seen == write_stats_25) { fprintf(stdout, "25%%..."); }
		//		else if (voxels_seen == write_stats_25 * size_t(2)) { fprintf(stdout, "50%%..."); }
		//		else if (voxels_seen == write_stats_25 * size_t(3)) { fprintf(stdout, "75%%..."); }
		//#endif

		const auto& morton = voxelArray[i];
		/*if (i == voxelArray.size() - 1)
		{
			std::cout << "last morton = " << morton << std::endl;
		}*/
		uint16_t x, y, z;
		morton3D_32_decode((morton & D_MORTON_32_FLAG), x, y, z);
		const Eigen::Vector3f nodeOrigin = modelBBox.min + width * Eigen::Vector3f((float)x, (float)y, (float)z);
		write_cube(nodeOrigin, Eigen::Vector3f(width, width, width), output, faceBegIdx);
	}
#ifndef SILENT
	//fprintf(stdout, "100%% \n");
#endif

	output.close();
}
